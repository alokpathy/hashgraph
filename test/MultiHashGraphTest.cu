#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "MultiHashGraph.cuh"

#include <algorithm>
#include <unistd.h>

// #define RAND_KEYS
// #define PRINT_KEYS

// #define BUILD_TEST

struct prg {
  hkey_t lo, hi;

  __host__ __device__ prg(hkey_t _lo=0, hkey_t _hi=0) : lo(_lo), hi(_hi) {};

  __host__ __device__ hkey_t operator()(index_t index) const {
    thrust::default_random_engine rng(index);
    thrust::uniform_int_distribution<hkey_t> dist(lo, hi);
    rng.discard(index);
    return dist(rng);
  }
};

// A recursive binary search function. It returns location of x in given array arr[l..r] is present, 
// otherwise it returns the bin id with the smallest value larger than x
int64_t binarySearch(hkey_t *bins, int32_t l, int64_t r, int32_t x) {
  if (r >= l) { 
    int64_t mid = l + (r - l) / 2; 
  
    // If the element is present at the middle itself 
    if (bins[mid] == x) 
        return mid; 

    // If element is smaller than mid, then it can only be present in left subarray 
    if (bins[mid] > x) 
        return binarySearch(bins, l, mid - 1, x); 

    // Else the element can only be present in right subarray 
    return binarySearch(bins, mid + 1, r, x); 
  } 
  
  // We reach here when element is not present in array and return the bin id 
  // of the smallest value greater than x
  return l; 
}

void enablePeerAccess(uint32_t gpuCount) {
  // Enable P2P access between each pair of GPUs.
  for (index_t j = 0; j < gpuCount; j++) {
    hipSetDevice(j);
    for (index_t i = 0; i < gpuCount; i++) {
      if (j != i) {
        int isCapable;
        hipDeviceCanAccessPeer(&isCapable, j, i);
        if (isCapable == 1) {
          hipError_t err = hipDeviceEnablePeerAccess(i, 0);
          if (err == hipErrorPeerAccessAlreadyEnabled) {
            hipGetLastError();
          }
        }
      }
    }
  }
} 

void generateInput(inputData *h_dVals, index_t countSize, index_t maxkey, uint32_t gpuCount,
                        index_t seed) {
  std::cout << "generating input" << std::endl;

  index_t avgKeyCount = std::ceil(countSize / ((double) gpuCount));
  for (index_t i = 0; i < gpuCount; i++) {
    hipSetDevice(i);

    index_t lo = avgKeyCount * i;
    index_t hi = avgKeyCount * (i + 1);
    hi = std::min(hi, countSize);

    index_t keyCount = hi - lo;

    hipMalloc(&h_dVals[i].d_keys, keyCount * sizeof(hkey_t));
    hipMalloc(&h_dVals[i].d_hash, keyCount * sizeof(HashKey));
    // RMM_ALLOC(&h_dVals[i].d_keys, keyCount * sizeof(hkey_t), 0);
    // RMM_ALLOC(&h_dVals[i].d_hash, keyCount * sizeof(HashKey), 0);

#ifdef RAND_KEYS
    // Randomly generate input keys on each device.
    thrust::counting_iterator<index_t> index_sequence_begin(seed);
    thrust::transform(thrust::device, index_sequence_begin, index_sequence_begin + keyCount,
                        h_dVals[i].d_keys, prg(0, maxkey - 1));
#else
    hkey_t *h_tmpKeys = new hkey_t[keyCount]();
    for (index_t j = lo; j < hi; j++) {
      h_tmpKeys[j - lo] = j;
    }
    hipMemcpy(h_dVals[i].d_keys, h_tmpKeys, keyCount * sizeof(hkey_t), hipMemcpyHostToDevice);
#endif

    h_dVals[i].len = keyCount;

#ifdef PRINT_KEYS
    std::cout << "keys gpu " << i << std::endl;
    thrust::device_ptr<hkey_t> td_keys = thrust::device_pointer_cast(h_dVals[i].d_keys);
    for (uint32_t j = 0; j < keyCount; j++) {
      std::cout << *(td_keys + j) << " ";
    }
    std::cout << std::endl;
#endif

    seed += keyCount;

  }
  std::cout << "done generating input" << std::endl;
}

int main(int argc, char **argv) {

  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);

  std::cout << "deviceCount: " << deviceCount << std::endl;

  char hostname[HOST_NAME_MAX];
  gethostname(hostname, HOST_NAME_MAX);

  std::cout << "hostname: " << hostname << std::endl;

  index_t countSizeA = 1L << 24;
  index_t maxkey = 1L << 26;

  uint32_t binCount = 16000;
  uint32_t gpuCount = 4;

  index_t lrbBins = -1;

  bool checkCorrectness = false;
  bool buildTest = false;

  index_t countSizeB = 1L << 22;

  if (argc >= 2 && argc < 9) {
    std::cerr << "Please specify all arguments.\n";
    return 1;
  }

  if (argc >= 3) {
    index_t size = strtoull(argv[1], NULL, 0);
    countSizeA = size;

    index_t key = strtoull(argv[2], NULL, 0);
    maxkey = key;

    binCount = atoi(argv[3]);
    gpuCount = atoi(argv[4]);

    lrbBins = strtoull(argv[5], NULL, 0);

    // char *correctnessFlag = atoi(argv[5]);
    // if (correctnessFlag > 0) {
    if (!strcmp(argv[6], "check")) {
      checkCorrectness = true;
    }
    
    countSizeB = strtoull(argv[7], NULL, 0);

    if (!strcmp(argv[8], "build")) {
      buildTest = true;
    }
  } 

  index_t tableSize = maxkey;

  std::cout << "countSizeA: " << countSizeA << std::endl;
  std::cout << "maxkey: " << maxkey << std::endl;

  // rmm_mgpu_context_t contextA;
  // rmm_mgpu_context_t contextB;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float buildTime = 0.0f; // milliseoncds

  enablePeerAccess(gpuCount);

  // rmmOptions_t rmmO;

  // rmmO.initial_pool_size = 1L << 60;
  // rmmO.allocation_mode = PoolAllocation;
  // rmmO.enable_logging = false;
  // rmmO.num_devices = 16;

  // int *devices = (int *)malloc(gpuCount * sizeof(int));
  // for (index_t i = 0; i < gpuCount; i++) {
  //   devices[i] = i;
  // }
  // 
  // rmmO.devices = devices;

  // rmmInitialize(&rmmO);


  if (buildTest) {
    inputData *h_dVals = new inputData[gpuCount]();
    generateInput(h_dVals, countSizeA, maxkey, gpuCount, 0);

    // MultiHashGraph mhg(h_dVals, countSizeA, maxkey, contextA, tableSize, binCount, lrbBins, gpuCount);
    MultiHashGraph mhg(h_dVals, countSizeA, maxkey, tableSize, binCount, lrbBins, gpuCount);

    omp_set_num_threads(gpuCount);

#ifdef HOST_PROFILE
    std::cout << "countBinSizes,countKeyBuff,populateKeyBuffs,countFinalKeys,allToAll,building,total"; // seconds
    std::cout << "times: ";
#else
    std::cout << "total_time\n"; // seconds
    std::cout << "times: ";
#endif

#ifdef CUDA_PROFILE
    hipProfilerStart();
#endif

    #pragma omp parallel
    {
      index_t tid = omp_get_thread_num();
      #pragma omp barrier
      if (tid == 0) {
        hipSetDevice(0);
        hipEventRecord(start);
      }
      

      mhg.build(true, tid);

      #pragma omp barrier
      if (tid == 0) {
        hipSetDevice(0);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&buildTime, start, stop);
      }

    } // pragma

    std::cout << (buildTime / 1000.0) << "\n"; // seconds

#ifdef CUDA_PROFILE
    hipProfilerStop();
    CHECK_ERROR("end of build");
#endif

    if (checkCorrectness) {
      mhg.destroyMulti();
      mhg.buildSingle();
    }
  } else {

    inputData *h_dValsA = new inputData[gpuCount]();
    inputData *h_dValsB = new inputData[gpuCount]();

    generateInput(h_dValsA, countSizeA, maxkey, gpuCount, 0);
    generateInput(h_dValsB, countSizeB, maxkey, gpuCount, countSizeA);

    // MultiHashGraph mhgA(h_dValsA, countSizeA, maxkey, contextA, tableSize, binCount, lrbBins, gpuCount);
    // MultiHashGraph mhgB(h_dValsB, countSizeB, maxkey, contextB, tableSize, binCount, lrbBins, gpuCount);
    MultiHashGraph mhgA(h_dValsA, countSizeA, maxkey, tableSize, binCount, lrbBins, gpuCount);
    MultiHashGraph mhgB(h_dValsB, countSizeB, maxkey, tableSize, binCount, lrbBins, gpuCount);

#ifdef MANAGED_MEM
    size_t size = 2 * (tableSize + gpuCount) * sizeof(index_t);
    hipMallocManaged(&mhgA.uvmPtrIntersect, size);
    // mhgA.prefixArrayIntersect = new index_t[gpuCount + 1]();
    mhgA.prefixArrayIntersect = new size_t[gpuCount + 1]();
    mhgA.totalSizeIntersect = size;
#endif

    keypair **h_dOutput = new keypair*[gpuCount]();
    index_t *h_Common = new index_t[gpuCount]();

    omp_set_num_threads(gpuCount);

#ifdef HOST_PROFILE
    std::cout << "countBinSizes,countKeyBuff,populateKeyBuffs,countFinalKeys,allToAll,"; // seconds
    std::cout << "countBinSizes,countKeyBuff,populateKeyBuffs,countFinalKeys,allToAll,building,outputalloc,intersect,total\n"; // seconds
    std::cout << "times: ";
#else
    std::cout << "total_time\n"; // seconds
    std::cout << "times: ";
#endif
#ifdef CUDA_PROFILE
    hipProfilerStart();
#endif

    hipSetDevice(0);
    hipEventRecord(start);

    #pragma omp parallel
    {
      index_t tid = omp_get_thread_num();
      mhgA.build(true, tid);

      #pragma omp master
      {
        mhgB.h_binSplits = mhgA.h_binSplits; // small memory leak.
        mhgB.h_dBinSplits = mhgA.h_dBinSplits;

#ifdef MANAGED_MEM
        mhgA.prefixArrayIntersect[0] = 0;
        for (index_t i = 1; i < gpuCount; i++) {
          index_t tidHashRange = mhgA.h_binSplits[i] - mhgA.h_binSplits[i - 1];
          index_t size = 2 * (tidHashRange + 1) * sizeof(index_t);
          mhgA.prefixArrayIntersect[i] = mhgA.prefixArrayIntersect[i - 1] + size;
        }
        mhgA.prefixArrayIntersect[gpuCount] = mhgA.totalSizeIntersect;

        mhgA.h_dCountCommon[0] = mhgA.uvmPtrIntersect;
        for (index_t i = 1; i < gpuCount; i++) {
          mhgA.h_dCountCommon[i] = mhgA.uvmPtrIntersect + 
                                        mhgA.prefixArrayIntersect[i];
        }
#endif
      } // master

      #pragma omp barrier

      mhgB.build(false, tid); // Build second HG but use same splits as first HG.

      #pragma omp barrier

      MultiHashGraph::intersect(mhgA, mhgB, h_Common, h_dOutput, tid);
    } // pragma

    hipSetDevice(0);
    hipEventRecord(stop);

#ifdef CUDA_PROFILE
    hipProfilerStop();
    CHECK_ERROR("end of intersect");
#endif
    
    hipEventSynchronize(stop);
    hipEventElapsedTime(&buildTime, start, stop);

    std::cout << (buildTime / 1000.0) << "\n"; // seconds

    if (checkCorrectness) {
      mhgA.buildSingle();
      mhgB.buildSingle();
      
      index_t outputSize = 0;
      for (index_t i = 0; i < gpuCount; i++) {
        outputSize += h_Common[i];
      }

      keypair *h_output = new keypair[outputSize]();
      index_t h_idx = 0;
      for (index_t i = 0; i < gpuCount; i++) {
        hipSetDevice(i);
        hipMemcpy(h_output + h_idx, h_dOutput[i], h_Common[i] * sizeof(keypair),
                            hipMemcpyDeviceToHost);
        h_idx += h_Common[i];
      }

      std::vector<hkey_t> result;
      result.reserve(outputSize);
      for (index_t i = 0; i < outputSize; i++) {
        result.push_back(h_output[i].right);
      }

      if (result.size() != result.capacity()) {
        std::cerr << "ERROR: RESULT ERROR" << std::endl;
        exit(0);
      }

      std::sort(mhgA.h_vals, mhgA.h_vals + countSizeA);
      std::sort(mhgB.h_vals, mhgB.h_vals + countSizeB);

      std::vector<hkey_t> ans;
      ans.reserve(outputSize);
      for (index_t i = 0; i < countSizeA; i++) {
        index_t ogIdx = binarySearch(mhgB.h_vals, 0, countSizeB - 1, mhgA.h_vals[i]);

        index_t idx = ogIdx;
        while (idx >= 0 && mhgB.h_vals[idx] == mhgA.h_vals[i]) {
          ans.push_back(mhgA.h_vals[i]);
          idx--;
        }

        idx = ogIdx + 1;
        while (idx < countSizeB && mhgB.h_vals[idx] == mhgA.h_vals[i]) {
          ans.push_back(mhgA.h_vals[i]);
          idx++;
        }
        // for (index_t j = 0; j < countSizeB; j++) {
        //   if (mhgA.h_vals[i] == mhgB.h_vals[j]) {
        //     ans.push_back(mhgA.h_vals[i]);
        //   }

        //   if (mhgA.h_vals[i] < mhgB.h_vals[j]) {
        //     break;
        //   }
        // } 
      }

      if (ans.size() != outputSize) {
        std::cerr << "ERROR: INTERSECT OUTPUT HAS INCORRECT SIZE" << std::endl;
        std::cerr << "ansSize: " << ans.size() << " outputSize: " << outputSize << std::endl;
        // exit(0);
      }

      std::sort(result.begin(), result.end());
      std::sort(ans.begin(), ans.end());

      if (result != ans) {
        std::cerr << "ERROR: INTERSECT OUTPUT HAS INCORRECT CONTENT" << std::endl;
        
        std::cout << "output: " << std::endl;
        for (auto i = result.begin(); i != result.end(); ++i) {
            std::cout << *i << " ";
        }
        std::cout << std::endl;

        std::cout << "ans: " << std::endl;
        for (auto i = ans.begin(); i != ans.end(); ++i) {
            std::cout << *i << " ";
        }
        std::cout << std::endl;

        exit(0);
      }
    }
  }
}
